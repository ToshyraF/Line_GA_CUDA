#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#define RANDOM_NUM ((float)rand()/(RAND_MAX))
#define CROSSOVER_RATE 0.7
#define LENGTH 8
#define N 100
struct Population
{
	int m,x,b;
	float FN; 
	float ratecross;
	float ratemutate;
};
int randomPop(){
	int pop=0;
	for(int i =0; i < LENGTH; i++){
		if(RANDOM_NUM > 0.5f){
			pop = pop<<1 | 1;
		}else{
			pop = pop << 1;
		}
	}
	return pop;
}
__device__ float fitness(Population pop,int y0){
	unsigned int y1,err;
	float FN;
	y1 = pop.x*pop.m + pop.b;
	err = y0 - y1;
	// err *= err;
	FN = 1+err;
	FN = 1/FN;
	return FN;

}

__global__ void calculate(Population *pop){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int y0 = pop[0].x*pop[0].m + pop[0].b;
	if(tid < N){
		pop[tid].FN = fitness(pop[tid],y0);
	}
}

__global__ void select(Population *pop,Population *out){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N){
		if(pop[tid].FN == 1){
			out[tid] = pop[tid];
			// count = count + 1;
		}
	}
}
__global__ void crossover(Population *pop){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N){
			int cross = (int)(pop[tid].ratecross*LENGTH);

			int forward=0;
			int invert=0;
			for(int i=0;i<cross;i++){
				forward = forward<<1 | 1;
			}
			invert = forward ^ 15;
			if(tid == N-1 || pop[tid].FN == 1){
				pop[tid] = pop[tid];
			}
			else{
				pop[tid].m = ((pop[tid].m&forward)|(pop[tid-1].m&invert));
				pop[tid].x = ((pop[tid].x&forward)|(pop[tid-1].x&invert));
				pop[tid].b = ((pop[tid].b&forward)|(pop[tid-1].b&invert));

			}
	}
}
__global__ void mutate(Population *pop){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N){
		int cross = (int)(pop[tid].ratemutate*LENGTH);
		int temp=1;
		for(int i=0;i<cross;i++){
			temp = temp<<1;
		}
		if(tid == N-1 || pop[tid].FN == 1){
				pop[tid] = pop[tid];
			}
		else{
			if(((pop[tid].m & temp) == 0)||((pop[tid].x & temp) == 0)||((pop[tid].m & temp) == 0)){
				pop[tid].m = pop[tid].m | temp;
				pop[tid].x = pop[tid].x | temp;
				pop[tid].b = pop[tid].b | temp;
			}
			else
			{
				pop[tid].m = pop[tid].m & (~temp & 15);
				pop[tid].x = pop[tid].x & (~temp & 15);
				pop[tid].b = pop[tid].b & (~temp & 15);
			}
			
		}
	}
}
int main(){
	srand((int)time(NULL));
	Population pop[N],out[N];

	Population *d_pop,*d_out;
	// int x[]={125,12,125,13,12,89};
	// int m[]={123,13,123,56,13,72};
	// int b[]={45,85,45,12,85,64};
	size_t size = N*sizeof(Population);
	//
	for(int i=0; i < N; i++){
         // pop[i].x = x[i];
         // pop[i].m = m[i];
         // pop[i].b = b[i];
		 pop[i].x = randomPop();
         pop[i].m = randomPop();
         pop[i].b = randomPop();
         pop[i].ratecross = RANDOM_NUM;
         pop[i].ratemutate = RANDOM_NUM;
	}

	for(int i=0;i<N;i++){
		printf("input m: %d x: %d b: %d \n",pop[i].m,pop[i].x,pop[i].b);
	}
	hipMalloc((void **)&d_pop,size);
	hipMalloc((void **)&d_out,size);

	hipMemcpy(d_pop,pop,size,hipMemcpyHostToDevice);
	for(int i=0;i<100000;i++){

		calculate<<<1,N>>>(d_pop);

		crossover<<<1,N>>>(d_pop);

		mutate<<<1,N>>>(d_pop);

		calculate<<<1,N>>>(d_pop);
	}
	hipMemcpy(out,d_pop,size,hipMemcpyDeviceToHost);

	for(int i=0;i<N;i++){
		printf("output m: %d x: %d b: %d FN:%f result: %d\n",out[i].m,out[i].x,out[i].b,out[i].FN,(out[i].m*out[i].x)+out[i].b);
		printf("option ratecross: %f ratemutate: %f\n",out[i].ratecross,out[i].ratemutate);
	}
	hipFree(d_pop);
		hipFree(d_out);
	return 0;
}